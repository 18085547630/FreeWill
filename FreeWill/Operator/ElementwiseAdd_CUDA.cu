#include "hip/hip_runtime.h"
#include "ElementwiseAdd_CUDA.h"
#include "../DeviceSelection.h"
#include <hip/hip_runtime.h>

template <typename DataType>
__global__ void elementwiseAdd(DataType *operandA, DataType *operandB, DataType rate, DataType *result, unsigned int size)
{
    int id = blockIdx.x*blockDim.x+threadIdx.x;
    if (id < size)
    {
        result[id] = operandA[id] + operandB[id] * rate;
    }
}
    
    
template <typename DataType>
__host__ void elementwiseAddCUDAKernel(DataType *operandA, DataType *operandB, DataType rate, DataType *result, unsigned int size)
{
    int blockSize = 1024;
    int gridSize =  size / blockSize ;

    if (size % blockSize != 0)
    {
        gridSize += 1;
    }

//    printf("gridsize:%d,%d",gridSize, blockSize);
    elementwiseAdd<DataType><<<gridSize, blockSize>>>(operandA, operandB, rate, result, size);
    CHECK_CUDA_ERROR
}

template __host__ void elementwiseAddCUDAKernel(float *operandA, float *operandB, float rate, float *result, unsigned int size);
template __host__ void elementwiseAddCUDAKernel(double *operandA, double *operandB, double rate, double *result, unsigned int size);
